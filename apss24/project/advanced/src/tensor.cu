#include "tensor.h"
#include <cstring>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

Tensor::Tensor(const vector<size_t> &shape_) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) { shape[i] = shape_[i]; }
    size_t N_ = num_elem();
    CHECK_CUDA(hipHostMalloc(&buf, N_ * sizeof(half_cpu))); // Pinned memory
    CHECK_CUDA(hipMalloc(&d_buf, N_ * sizeof(half)));
}

Tensor::Tensor(const vector<size_t> &shape_, half_cpu *buf_) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) { shape[i] = shape_[i]; }
    size_t N_ = num_elem();
    CHECK_CUDA(hipHostMalloc(&buf, N_ * sizeof(half_cpu))); // Pinned memory
    memcpy(buf, buf_, N_ * sizeof(half_cpu));
    CHECK_CUDA(hipMalloc(&d_buf, N_ * sizeof(half)));
    to_device_async(nullptr); // Asynchronous transfer
}

Tensor::~Tensor() {
    if (buf != nullptr) CHECK_CUDA(hipHostFree(buf)); // Free pinned memory
    if (d_buf != nullptr) CHECK_CUDA(hipFree(d_buf));
}

size_t Tensor::num_elem() {
    size_t size = 1;
    for (size_t i = 0; i < ndim; i++) { size *= shape[i]; }
    return size;
}

void Tensor::to_device_async(hipStream_t stream) {
    size_t N_ = num_elem();
    CHECK_CUDA(hipMemcpyAsync(d_buf, buf, N_ * sizeof(half), hipMemcpyHostToDevice, stream));
}

void Tensor::to_host_async(hipStream_t stream) {
    size_t N_ = num_elem();
    CHECK_CUDA(hipMemcpyAsync(buf, d_buf, N_ * sizeof(half), hipMemcpyDeviceToHost, stream));
}

// Synchronous versions (for backwards compatibility)
void Tensor::to_device() {
    to_device_async(nullptr);
    CHECK_CUDA(hipDeviceSynchronize());
}

void Tensor::to_host() {
    to_host_async(nullptr);
    CHECK_CUDA(hipDeviceSynchronize());
}