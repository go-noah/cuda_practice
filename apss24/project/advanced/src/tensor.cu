#include "tensor.h"
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

Tensor::Tensor(const vector<size_t> &shape_, bool gpu) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) { shape[i] = shape_[i]; }
    size_t N_ = num_elem();
    is_gpu = gpu;
    if (is_gpu) {
        CHECK_CUDA(hipMalloc(&buf, N_ * sizeof(half)));
    } else {
        buf = (half *) calloc(N_, sizeof(half));
    }
}

Tensor::Tensor(const vector<size_t> &shape_, half_cpu *buf_, bool gpu) {
    ndim = shape_.size();
    for (size_t i = 0; i < ndim; i++) { shape[i] = shape_[i]; }
    size_t N_ = num_elem();
    is_gpu = gpu;
    if (is_gpu) {
        CHECK_CUDA(hipMalloc(&buf, N_ * sizeof(half)));
        CHECK_CUDA(hipMemcpy(buf, buf_, N_ * sizeof(half), hipMemcpyHostToDevice));
    } else {
        buf = (half *) malloc(N_ * sizeof(half));
        memcpy(buf, buf_, N_ * sizeof(half));
    }
}

Tensor::~Tensor() {
    if (is_gpu) {
        if (buf != nullptr) CHECK_CUDA(hipFree(buf));
    } else {
        if (buf != nullptr) free(buf);
    }
}

void Tensor::to_gpu() {
    if (!is_gpu) {
        size_t N_ = num_elem();
        half *gpu_buf;
        CHECK_CUDA(hipMalloc(&gpu_buf, N_ * sizeof(half)));
        CHECK_CUDA(hipMemcpy(gpu_buf, buf, N_ * sizeof(half), hipMemcpyHostToDevice));
        free(buf);
        buf = gpu_buf;
        is_gpu = true;
    }
}

void Tensor::to_cpu() {
    if (is_gpu) {
        size_t N_ = num_elem();
        half *cpu_buf = (half *) malloc(N_ * sizeof(half));
        CHECK_CUDA(hipMemcpy(cpu_buf, buf, N_ * sizeof(half), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipFree(buf));
        buf = cpu_buf;
        is_gpu = false;
    }
}

size_t Tensor::num_elem() {
    size_t size = 1;
    for (size_t i = 0; i < ndim; i++) { size *= shape[i]; }
    return size;
}